#include "hip/hip_runtime.h"
/*
* Copyright (c) Huawei Technologies Co., Ltd. 2019-2019. All rights reserved.
*
* This program is free software; you can redistribute it and/or modify
* it under the terms of the Apache License Version 2.0.You may not use this file except in compliance with the License.
*
* This program is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
* Apache License for more details at
* http://www.apache.org/licenses/LICENSE-2.0
*
* @brief fake_dequant_quant custom op kernel func CUDA implement
*
* @file dequant_quant_impl.cpp
*
* @version 1.0
*/
#include "dequant_quant.h"
#include <cmath>
#include <mutex>
#include <cfloat>
#include <algorithm>
#include <memory>
#include <numeric>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include "util.h"
#include "cast_util.h"
#include "cast.cuh"

using namespace util;
#define FLOAT_TYPE_ID (1)
#define INT8_TYPE_ID (3)
#define FLOAT16_TYPE_ID (10)

__global__ void FakeQuantCudaImpl(
    const float* inputData,
    float* outputData,
    int64_t length,
    int64_t bound,
    FakeCalParams calParams)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int dataTmp = 0;
    if (tid < length) {
        if (calParams.fakePrecisonMode == util::FORCE_FP16_QUANT) {
            dataTmp = static_cast<int>(rint(CastToFP16Precision(
                CastToFP16Precision(CastToFP16Precision(inputData[tid]) * FakeFp16PrecisionData(calParams.scale)) + calParams.offset)));
        } else {
            dataTmp = static_cast<int>(rint(inputData[tid] * calParams.scale)) + calParams.offset;
        }
        dataTmp = (dataTmp < -bound) ? -bound : dataTmp;
        dataTmp = (dataTmp > bound - 1) ? bound - 1 : dataTmp;
        outputData[tid] = static_cast<float>((dataTmp - calParams.offset));
    }
}


__global__ void FakeQuantInt8CudaImpl(
    const float* inputData,
    int8_t* outputData,
    int64_t length,
    int64_t bound,
    FakeCalParams calParams)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int dataTmp = 0;
    if (tid < length) {
        if (calParams.fakePrecisonMode == util::FORCE_FP16_QUANT) {
            if ((calParams.scale > MAX_FP16) || (calParams.scale > 0 && calParams.scale < MIN_FP16)) {
                calParams.scale = CastToFP16Precision(sqrt(calParams.scale)) * CastToFP16Precision(sqrt(calParams.scale));
            } else {
                calParams.scale = CastToFP16Precision(calParams.scale);
            }
            dataTmp = static_cast<int>(rint(CastToFP16Precision(CastToFP16Precision(inputData[tid]) * calParams.scale))) 
                + calParams.offset;
        } else {
            dataTmp = static_cast<int>(rint(inputData[tid] * calParams.scale)) + calParams.offset;
        }
        dataTmp = (dataTmp < -bound) ? -bound : dataTmp;
        dataTmp = (dataTmp > bound - 1) ? bound - 1 : dataTmp;
        outputData[tid] = static_cast<float>(dataTmp);
    }
}


__global__ void FakeDeQuantCudaImpl(
    const float* inputData,
    float* outputData,
    int64_t length,
    struct DequantParam dequantParam,
    int64_t fakePrecisionMode)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < length) {
        int channelIndex = !dequantParam.channelWise ? 0 : (tid % (dequantParam.chwSize)) / dequantParam.hwSize;
        unsigned int shiftBit = (dequantParam.paramData[channelIndex] & dequantParam.shiftnMask) >> 32;
        if (shiftBit != 0 ) {
            dequantParam.clipMode = CLIP_16;
        }
        float clipMin = -static_cast<float>(pow(BINARY_BASE, dequantParam.clipMode - 1));
        float clipMax = static_cast<float>(pow(BINARY_BASE, dequantParam.clipMode - 1) - 1);
        float shiftValue = pow(BINARY_BASE, shiftBit);
        unsigned int deqscaleUint = dequantParam.paramData[channelIndex] & dequantParam.deqscaleMask;
        float* deqscalePtr = reinterpret_cast<float*>(&deqscaleUint);

        float tmpData = 0;
        if (std::fabs(shiftValue - 1) <= FLT_EPSILON) {
            tmpData = inputData[tid];
        } else {
            tmpData = floor(inputData[tid] / shiftValue);
        }
        tmpData = (tmpData < clipMin) ? clipMin : tmpData;
        tmpData = (tmpData > clipMax) ? clipMax : tmpData;
        if (fakePrecisionMode == util::FORCE_FP16_QUANT) {
            outputData[tid] = CastToFP16Precision(tmpData * CastToS19(*deqscalePtr) * shiftValue);
        } else {
            outputData[tid] = tmpData * (*deqscalePtr) * shiftValue;
        }
    }
}


__global__ void FakeAntiQuantCudaImpl(
    const float* inputData,
    float* outputData,
    int64_t length,
    float scaleData)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < length) {
        outputData[tid] = inputData[tid] * scaleData;
    }
}

inline int checkCudaError(hipError_t error, const char *file, const int line)
{
    if (error != hipSuccess) {
        std::cerr << "CUDA CALL FAILED:" << file << "( " << line << ")- " << hipGetErrorString(error) << std::endl;
        return AmctCommon::CUDA_ASYNC_ERROR;
    }
    return AmctCommon::SUCCESS;
}

int FakeQuantCuda(InputDataParam param,
    int64_t quantBits,
    float scale,
    int64_t offset)
{
    FakeCalParams calParams = {param.fakePrecisionMode, scale, offset};
    int64_t bound = pow(2, quantBits - 1);
    hipError_t errAsync = hipDeviceSynchronize();
    checkCudaError(errAsync, __FILE__, __LINE__);

    thrust::device_vector<float> inCast;
    if (param.inType == FLOAT16_TYPE_ID) {
        inCast.resize(param.length);
        float* inSavePtr = thrust::raw_pointer_cast(inCast.data());
        DataCastToFloat32Functor<util::GPUDevice, uint16_t>()(reinterpret_cast<const uint16_t*>(param.in), inSavePtr,
            param.length);
        if (param.outType == INT8_TYPE_ID) {
            // in_16, out_8
            FakeQuantInt8CudaImpl<<<AMCT_GET_BLOCKS(param.length), AMCT_CUDA_NUM_THREADS>>>(
                inSavePtr, reinterpret_cast<int8_t*>(param.out), param.length, bound, calParams);
        } else if (param.outType == FLOAT16_TYPE_ID) {
            // in_16, out_16
            thrust::device_vector<float> outCast(param.length, 0);
            float* outSavePtr = thrust::raw_pointer_cast(outCast.data());
            FakeQuantCudaImpl<<<AMCT_GET_BLOCKS(param.length), AMCT_CUDA_NUM_THREADS>>>(
                inSavePtr, outSavePtr, param.length, bound, calParams);
            DataCastToFloat16Functor<util::GPUDevice, float>()(outSavePtr, reinterpret_cast<uint16_t*>(param.out),
                param.length);
        } else {
            // in_16, out_32
            FakeQuantCudaImpl<<<AMCT_GET_BLOCKS(param.length), AMCT_CUDA_NUM_THREADS>>>(
                inSavePtr, reinterpret_cast<float*>(param.out), param.length, bound, calParams);
        }
        errAsync = hipDeviceSynchronize();
        checkCudaError(errAsync, __FILE__, __LINE__);
        if (errAsync != hipSuccess) {
            return AmctCommon::CUDA_ERROR;
        }
        return AmctCommon::SUCCESS;
    }

    if (param.outType == INT8_TYPE_ID) {
        // in_32, out_8
        FakeQuantInt8CudaImpl<<<AMCT_GET_BLOCKS(param.length), AMCT_CUDA_NUM_THREADS>>>(
            reinterpret_cast<const float*>(param.in), reinterpret_cast<int8_t*>(param.out),
            param.length, bound, calParams);
    } else {
        // in_32, out_32
        FakeQuantCudaImpl<<<AMCT_GET_BLOCKS(param.length), AMCT_CUDA_NUM_THREADS>>>(
            reinterpret_cast<const float*>(param.in), reinterpret_cast<float*>(param.out),
            param.length, bound, calParams);
    }

    errAsync = hipDeviceSynchronize();
    checkCudaError(errAsync, __FILE__, __LINE__);
    if (errAsync != hipSuccess) {
        return AmctCommon::CUDA_ERROR;
    }
    return AmctCommon::SUCCESS;
}


int FakeDequantCuda(InputDataParam param, DequantParam dequantParam)
{
    dequantParam.shiftnMask = 0x000000ff00000000;
    dequantParam.deqscaleMask = 0x00000000ffffffff;
    hipError_t errAsync = hipDeviceSynchronize();
    checkCudaError(errAsync, __FILE__, __LINE__);
    if (errAsync != hipSuccess) {
        return AmctCommon::CUDA_ERROR;
    }
    if (param.outType != 1) {
        thrust::device_vector<float> outCast(param.length, 0);
        float* outSavePtr = thrust::raw_pointer_cast(outCast.data());
        // in_16, out_16
        if (param.inType != 1) {
            thrust::device_vector<float> inCast(param.length, 0);
            float* inSavePtr = thrust::raw_pointer_cast(inCast.data());
            DataCastToFloat32Functor<util::GPUDevice, uint16_t>()(reinterpret_cast<const uint16_t*>(param.in), inSavePtr, param.length);
            FakeDeQuantCudaImpl<<<AMCT_GET_BLOCKS(param.length), AMCT_CUDA_NUM_THREADS, 0, 0>>>(
                inSavePtr, outSavePtr, param.length, dequantParam, param.fakePrecisionMode);
        } else {
            // in_32, out_16
            FakeDeQuantCudaImpl<<<AMCT_GET_BLOCKS(param.length), AMCT_CUDA_NUM_THREADS, 0, 0>>>(
                reinterpret_cast<const float*>(param.in), outSavePtr, param.length, dequantParam, param.fakePrecisionMode);
        }
        DataCastToFloat16Functor<util::GPUDevice, float>()(outSavePtr, reinterpret_cast<uint16_t*>(param.out), param.length);
        errAsync = hipDeviceSynchronize();
        checkCudaError(errAsync, __FILE__, __LINE__);
        if (errAsync != hipSuccess) {
            return AmctCommon::CUDA_ERROR;
        }
        return AmctCommon::SUCCESS;
    }
    // in_32, out_32
    FakeDeQuantCudaImpl<<<AMCT_GET_BLOCKS(param.length), AMCT_CUDA_NUM_THREADS, 0, 0>>>(
        reinterpret_cast<const float*>(param.in), reinterpret_cast<float*>(param.out),
        param.length, dequantParam, param.fakePrecisionMode);
    errAsync = hipDeviceSynchronize();
    checkCudaError(errAsync, __FILE__, __LINE__);
    if (errAsync != hipSuccess) {
        return AmctCommon::CUDA_ERROR;
    }
    return AmctCommon::SUCCESS;
}


int FakeAntiQuantCuda(InputDataParam param, float scaleData)
{
    hipError_t errAsync;
    if (param.outType != 1) {
        thrust::device_vector<float> outCast(param.length, 0);
        float* outSavePtr = thrust::raw_pointer_cast(outCast.data());
        // in_16, out_16
        if (param.inType != 1) {
            thrust::device_vector<float> inCast(param.length, 0);
            float* inSavePtr = thrust::raw_pointer_cast(inCast.data());
            DataCastToFloat32Functor<util::GPUDevice, uint16_t>()(reinterpret_cast<const uint16_t*>(param.in), inSavePtr, param.length);
            FakeAntiQuantCudaImpl<<<AMCT_GET_BLOCKS(param.length), AMCT_CUDA_NUM_THREADS>>>(
                inSavePtr, outSavePtr, param.length, scaleData);
        } else {
            // in_32, out_16
            FakeAntiQuantCudaImpl<<<AMCT_GET_BLOCKS(param.length), AMCT_CUDA_NUM_THREADS>>>(
                reinterpret_cast<const float*>(param.in), outSavePtr, param.length, scaleData);
        }
        DataCastToFloat16Functor<util::GPUDevice, float>()(outSavePtr, reinterpret_cast<uint16_t*>(param.out), param.length);
        errAsync = hipDeviceSynchronize();
        checkCudaError(errAsync, __FILE__, __LINE__);
        if (errAsync != hipSuccess) {
            return AmctCommon::CUDA_ERROR;
        }
        return AmctCommon::SUCCESS;
    }
    // in_32, out_32
    FakeAntiQuantCudaImpl<<<AMCT_GET_BLOCKS(param.length), AMCT_CUDA_NUM_THREADS>>>(
        reinterpret_cast<const float*>(param.in), reinterpret_cast<float*>(param.out),
        param.length, scaleData);
    errAsync = hipDeviceSynchronize();
    checkCudaError(errAsync, __FILE__, __LINE__);
    if (errAsync != hipSuccess) {
        return AmctCommon::CUDA_ERROR;
    }
    return AmctCommon::SUCCESS;
}